#include "hip/hip_runtime.h"
#define _CRT_SECURE_NO_DEPRECATE
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <string.h>
#include <sys/timeb.h>
#include "hip/hip_runtime.h"
#include ""

#include "tinycthread.c"
#include "util.c"

__constant__ unsigned char const_colormap[(MAX_ITERATION + 1) * 3];

hipError_t generate_img(unsigned char* image);// , unsigned char* colormap);


__device__ void set_pixel(unsigned char* image, int width, int x, int y, unsigned char* c) {
	image[4 * width * y + 4 * x + 0] = c[0];
	image[4 * width * y + 4 * x + 1] = c[1];
	image[4 * width * y + 4 * x + 2] = c[2];
	image[4 * width * y + 4 * x + 3] = 255;
}

/* This should be conveted into a GPU kernel */
__global__ void generate_image(unsigned char* image, int const_width, int const_height, int const_max) {

	int row, col, index, iteration;
	double c_re, c_im, x, y, x_new;

	index = threadIdx.x + blockIdx.x * blockDim.x;

	while (index < const_width * const_height) {

		row = index / const_width;
		col = index % const_width;

		c_re = (col - const_width / 2.0) * 4.0 / const_width;
		c_im = (row - const_height / 2.0) * 4.0 / const_width;
		x = 0, y = 0;
		iteration = 0;
		while (x * x + y * y <= 4 && iteration < const_max) {
			x_new = x * x - y * y + c_re;
			y = 2 * x * y + c_im;
			x = x_new;
			iteration++;
		}
		if (iteration > const_max) {
			iteration = const_max;
		}
		set_pixel(image, const_width, col, row, &const_colormap[iteration * 3]);
		index += blockDim.x * gridDim.x;
	}
}

int main(int argc, char** argv) {
	double times[REPEAT];
	struct timeb start, end;
	int r;
	char path[] = "./";

	unsigned char* colormap = (unsigned char*)malloc((MAX_ITERATION + 1) * 3);
	unsigned char* image = (unsigned char*)malloc(WIDTH * HEIGHT * 4);

	hipError_t cudaStatus;

	init_colormap(MAX_ITERATION, colormap);

	cudaStatus = hipMemcpyToSymbol(HIP_SYMBOL(const_colormap), colormap, (MAX_ITERATION + 1) * 3, 0, hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "Copy colormap to constant memory from device to host failed!");
		free(image);
		free(colormap);
		return 1;
	}
	free(colormap);


	for (r = 0; r < REPEAT; r++) {
		memset(image, 0, WIDTH * HEIGHT * 4);

		ftime(&start);

		cudaStatus = generate_img(image); // , colormap);
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "generate_img failed!");
			free(image);
			free(colormap);
			return 1;
		}

		ftime(&end);
		times[r] = end.time - start.time + ((double)end.millitm - (double)start.millitm) / 1000.0;

		sprintf(path, IMAGE, "gpu", r);
		save_image(path, image, WIDTH, HEIGHT);
		progress("gpu", r, times[r]);
	}
	report("gpu", times);

	free(image);
	//free(colormap);

	//hipDeviceReset must be  called before exiting in order for profiling and
	// tracing tools such as Nsight and Visual Profiler to show complete traces.
	cudaStatus = hipDeviceReset();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipDeviceReset failed!");
		return 1;
	}
	return 0;
}


hipError_t generate_img(unsigned char* image){ //, unsigned char* colormap) {

	unsigned char* dev_image;// , * dev_colormap;
	hipError_t cudaStatus;


	cudaStatus = hipSetDevice(0);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipSetDevice failed!");
		goto Error;
	}

	// Allocate GPU buffer for image
	/*cudaStatus = hipMalloc((void**)&dev_colormap, (MAX_ITERATION + 1) * 3);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		goto Error;
	}*/

	cudaStatus = hipMalloc((void**)&dev_image, WIDTH * HEIGHT * 4);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		goto Error;
	}


	/*cudaStatus = hipMemcpy(dev_colormap, colormap, (MAX_ITERATION + 1) * 3, hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
		goto Error;
	}*/

	generate_image<<<BLOCKS, THREADS>>>(dev_image, WIDTH, HEIGHT, MAX_ITERATION);

	// Errors when lunching the kernel
	cudaStatus = hipGetLastError();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "Failed to generate image: %s\n", hipGetErrorString(cudaStatus));
		goto Error;
	}

	// hipDeviceSynchronize waits for the kernel to finish and returns
	// any errors encountered during the launch.
	cudaStatus = hipDeviceSynchronize();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching generate_image!\n", cudaStatus);
		goto Error;
	}

	// Copy image from GPU buffer to host memory
	cudaStatus = hipMemcpy(image, dev_image, WIDTH * HEIGHT * 4, hipMemcpyDeviceToHost);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "Copy image from device to host failed!");
		goto Error;
	}

Error:
	hipFree(dev_image);
	//hipFree(dev_colormap);
	return cudaStatus;
}
