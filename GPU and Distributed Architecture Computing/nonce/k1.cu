//
// Created by Katy on 08-Dec-21.
//

#include <stdio.h>
#include <vector>
#include <hip/hip_runtime.h>

using namespace std;

__device__ void lock(int *mutex) {
    while (atomicCAS(mutex, 0, 1) != 0);
}

__device__ void unlock(int *mutex) {
    atomicExch(mutex, 0);
}

//__global__ void ck_test_lock(int *, int*);
//__global__ void ck_test_lock_2(int *, int*);
__global__ void ck_test_lock_3(int *, int *);
//__global__ void ck_test_lock_4(int *, int *);

int *mutex;

int test_cuda_mutex() {
    hipMalloc(&mutex, sizeof(int));
    hipMemset(mutex, 0, sizeof(int));
    int *status;
    hipMallocManaged(&status, sizeof(int));
    *status = 0;

    // This one can pass since it contains 2 blocks of threads
    // ck_test_lock<<<2, 1>>>(status, mutex);
    // cudaDeviceSynchronize();

    // These two will not pass; deadlock between threads in a warp
    // ck_test_lock<<<1, 2>>>(status, mutex);
    // ck_test_lock<<<1, 32>>>(status, mutex);
    // cudaDeviceSynchronize();

    // This one can pass since it contains 2 blocks of threads
    // ck_test_lock_2<<<2, 1>>>(status, mutex);
    // ck_test_lock_2<<<1, 2>>>(status, mutex);
    // cudaDeviceSynchronize();

    // This works well
    ck_test_lock_3<<<64, 1024>>>(status, mutex);
    // This works badly
    // ck_test_lock_4<<<1, 32>>>(status, mutex);

    if (hipSuccess != hipGetLastError())
        printf("Error!\n");
    hipDeviceSynchronize();
    printf("result is %d\n\n\n", *status);
    hipFree(status);
    return 0;
}

//__global__ void ck_test_lock_4(int *status, int *mutex) {
//    bool leave = true;
//    while (leave) {
//        if (atomicCAS(mutex, 0, 1) == 0) {
//            status[0] = threadIdx.x;
//            leave = false;
//            atomicExch(mutex, 0);
//            break;
//        }
//    }
//}

__global__ void ck_test_lock_3(int *status, int *mutex) {
    bool leave = true;
    while (!*status) {
        if (atomicCAS(mutex, 0, 1) == 0) {
            *status = 1;
//            leave = false;
            atomicExch(mutex, 0);
        }
//        break;
    }
}

//__global__ void ck_test_lock_2(int *status, int *mutex) {
//    int c = 0;
//    LOCKFRONT:
//    if (c > 10) {
//        // thread 1 runs 10 times; go out
//        // thread 0 will start to work.
//        goto EXIT;
//    }
//    printf("blkID = %d, trdID = %d; mutex = %d\n", blockIdx.x, threadIdx.x, *mutex);
//    if (atomicCAS(mutex, 0, 1) == 0) {
//        // thread 0 is blocked here
//        printf("%d: I start the first line  =====================\n", threadIdx.x);
//        status[0] = threadIdx.x;
//        atomicExch(mutex, 0);
//    } else {
//        // thread 1 goes here
//        c++;
//        printf("%d %d %d\n", threadIdx.x, c, *mutex);
//        goto LOCKFRONT;
//    }
//    EXIT:
//}
//
//__global__ void ck_test_lock(int *status, int *mutex) {
//    printf("blkID = %d, trdID = %d; mutex = %d\n", blockIdx.x, threadIdx.x, *mutex);
//    lock(mutex);
//    printf("============mutex is %d ========\n", *mutex);
//    status[0] = threadIdx.x;
//    unlock(mutex);
//}

int main() {
    for (int T = 0; T < 1; T++) {
        test_cuda_mutex();
    }
}