#include "hip/hip_runtime.h"
#define _CRT_SECURE_NO_DEPRECATE

#include <string.h>
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <sys/timeb.h>
#include "common.h"

/*
 * SHA-1 GPU implementation
 */
typedef struct {
    unsigned long total[2];     /* number of bytes processed  */
    unsigned long state[5];     /* intermediate digest state  */
    unsigned char buffer[64];   /* data block being processed */
} sha1_gpu_context;


__device__ static const unsigned char sha1_padding[64] =
        {
                0x80, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0,
                0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0,
                0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0,
                0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0
        };


/*
 * Prepare SHA-1 for execution.
 */
__device__ void sha1_gpu_starts(sha1_gpu_context *ctx) {
    ctx->total[0] = 0;
    ctx->total[1] = 0;
    ctx->state[0] = 0x67452301;
    ctx->state[1] = 0xEFCDAB89;
    ctx->state[2] = 0x98BADCFE;
    ctx->state[3] = 0x10325476;
    ctx->state[4] = 0xC3D2E1F0;
}


/*
 * Process one block of data.
 */
__device__ static void sha1_gpu_process(sha1_gpu_context *ctx, unsigned char data[64]) {
    unsigned long temp, W[16] = {0,}, A, B, C, D, E;

    GET_UINT32_BE(W[0], data, 0);
    GET_UINT32_BE(W[1], data, 4);
    GET_UINT32_BE(W[2], data, 8);
    GET_UINT32_BE(W[3], data, 12);
    GET_UINT32_BE(W[4], data, 16);
    GET_UINT32_BE(W[5], data, 20);
    GET_UINT32_BE(W[6], data, 24);
    GET_UINT32_BE(W[7], data, 28);
    GET_UINT32_BE(W[8], data, 32);
    GET_UINT32_BE(W[9], data, 36);
    GET_UINT32_BE(W[10], data, 40);
    GET_UINT32_BE(W[11], data, 44);
    GET_UINT32_BE(W[12], data, 48);
    GET_UINT32_BE(W[13], data, 52);
    GET_UINT32_BE(W[14], data, 56);
    GET_UINT32_BE(W[15], data, 60);

#define S(x, n) ((x << n) | ((x & 0xFFFFFFFF) >> (32 - n)))

#define R(t)                        \
(                                                       \
    temp = W[(t -  3) & 0x0F] ^ W[(t - 8) & 0x0F] ^     \
           W[(t - 14) & 0x0F] ^ W[ t      & 0x0F],      \
    ( W[t & 0x0F] = S(temp,1) )                         \
)

#define P(a, b, c, d, e, x)                                  \
{                                                       \
    e += S(a,5) + F(b,c,d) + K + x; b = S(b,30);        \
}

    A = ctx->state[0];
    B = ctx->state[1];
    C = ctx->state[2];
    D = ctx->state[3];
    E = ctx->state[4];

#define F(x, y, z) (z ^ (x & (y ^ z)))
#define K 0x5A827999

    P(A, B, C, D, E, W[0]);
    P(E, A, B, C, D, W[1]);
    P(D, E, A, B, C, W[2]);
    P(C, D, E, A, B, W[3]);
    P(B, C, D, E, A, W[4]);
    P(A, B, C, D, E, W[5]);
    P(E, A, B, C, D, W[6]);
    P(D, E, A, B, C, W[7]);
    P(C, D, E, A, B, W[8]);
    P(B, C, D, E, A, W[9]);
    P(A, B, C, D, E, W[10]);
    P(E, A, B, C, D, W[11]);
    P(D, E, A, B, C, W[12]);
    P(C, D, E, A, B, W[13]);
    P(B, C, D, E, A, W[14]);
    P(A, B, C, D, E, W[15]);
    P(E, A, B, C, D, R(16));
    P(D, E, A, B, C, R(17));
    P(C, D, E, A, B, R(18));
    P(B, C, D, E, A, R(19));

#undef K
#undef F

#define F(x, y, z) (x ^ y ^ z)
#define K 0x6ED9EBA1

    P(A, B, C, D, E, R(20));
    P(E, A, B, C, D, R(21));
    P(D, E, A, B, C, R(22));
    P(C, D, E, A, B, R(23));
    P(B, C, D, E, A, R(24));
    P(A, B, C, D, E, R(25));
    P(E, A, B, C, D, R(26));
    P(D, E, A, B, C, R(27));
    P(C, D, E, A, B, R(28));
    P(B, C, D, E, A, R(29));
    P(A, B, C, D, E, R(30));
    P(E, A, B, C, D, R(31));
    P(D, E, A, B, C, R(32));
    P(C, D, E, A, B, R(33));
    P(B, C, D, E, A, R(34));
    P(A, B, C, D, E, R(35));
    P(E, A, B, C, D, R(36));
    P(D, E, A, B, C, R(37));
    P(C, D, E, A, B, R(38));
    P(B, C, D, E, A, R(39));

#undef K
#undef F

#define F(x, y, z) ((x & y) | (z & (x | y)))
#define K 0x8F1BBCDC

    P(A, B, C, D, E, R(40));
    P(E, A, B, C, D, R(41));
    P(D, E, A, B, C, R(42));
    P(C, D, E, A, B, R(43));
    P(B, C, D, E, A, R(44));
    P(A, B, C, D, E, R(45));
    P(E, A, B, C, D, R(46));
    P(D, E, A, B, C, R(47));
    P(C, D, E, A, B, R(48));
    P(B, C, D, E, A, R(49));
    P(A, B, C, D, E, R(50));
    P(E, A, B, C, D, R(51));
    P(D, E, A, B, C, R(52));
    P(C, D, E, A, B, R(53));
    P(B, C, D, E, A, R(54));
    P(A, B, C, D, E, R(55));
    P(E, A, B, C, D, R(56));
    P(D, E, A, B, C, R(57));
    P(C, D, E, A, B, R(58));
    P(B, C, D, E, A, R(59));

#undef K
#undef F

#define F(x, y, z) (x ^ y ^ z)
#define K 0xCA62C1D6

    P(A, B, C, D, E, R(60));
    P(E, A, B, C, D, R(61));
    P(D, E, A, B, C, R(62));
    P(C, D, E, A, B, R(63));
    P(B, C, D, E, A, R(64));
    P(A, B, C, D, E, R(65));
    P(E, A, B, C, D, R(66));
    P(D, E, A, B, C, R(67));
    P(C, D, E, A, B, R(68));
    P(B, C, D, E, A, R(69));
    P(A, B, C, D, E, R(70));
    P(E, A, B, C, D, R(71));
    P(D, E, A, B, C, R(72));
    P(C, D, E, A, B, R(73));
    P(B, C, D, E, A, R(74));
    P(A, B, C, D, E, R(75));
    P(E, A, B, C, D, R(76));
    P(D, E, A, B, C, R(77));
    P(C, D, E, A, B, R(78));
    P(B, C, D, E, A, R(79));

#undef K
#undef F

    ctx->state[0] += A;
    ctx->state[1] += B;
    ctx->state[2] += C;
    ctx->state[3] += D;
    ctx->state[4] += E;
}


/*
 * Splits input message into blocks and processes them one by one. Also
 * checks how many 0 need to be padded and processes the last, padded, block.
 */
__device__ void sha1_gpu_update(sha1_gpu_context *ctx, unsigned char *input, int ilen) {
    int fill;
    unsigned long left;

    if (ilen <= 0)
        return;

    left = ctx->total[0] & 0x3F;
    fill = 64 - left;

    ctx->total[0] += ilen;
    ctx->total[0] &= 0xFFFFFFFF;

    if (ctx->total[0] < (unsigned long) ilen)
        ctx->total[1]++;

    if (left && ilen >= fill) {
        memcpy((void *) (ctx->buffer + left), (void *) input, fill);
        sha1_gpu_process(ctx, ctx->buffer);
        input += fill;
        ilen -= fill;
        left = 0;
    }

    while (ilen >= 64) {
        sha1_gpu_process(ctx, input);
        input += 64;
        ilen -= 64;
    }

    if (ilen > 0) {
        memcpy((void *) (ctx->buffer + left), (void *) input, ilen);
    }
}


/*
 * Process padded block and return hash to user.
 */
__device__ void sha1_gpu_finish(sha1_gpu_context *ctx, unsigned char *output) {
    unsigned long last, padn;
    unsigned long high, low;
    unsigned char msglen[8];


    high = (ctx->total[0] >> 29) | (ctx->total[1] << 3);
    low = (ctx->total[0] << 3);

    PUT_UINT32_BE(high, msglen, 0);
    PUT_UINT32_BE(low, msglen, 4);

    last = ctx->total[0] & 0x3F;
    padn = (last < 56) ? (56 - last) : (120 - last);

    sha1_gpu_update(ctx, (unsigned char *) sha1_padding, padn);
    sha1_gpu_update(ctx, msglen, 8);

    PUT_UINT32_BE(ctx->state[0], output, 0);
    PUT_UINT32_BE(ctx->state[1], output, 4);
    PUT_UINT32_BE(ctx->state[2], output, 8);
    PUT_UINT32_BE(ctx->state[3], output, 12);
    PUT_UINT32_BE(ctx->state[4], output, 16);
}

__constant__  unsigned char const_string[1024];
__constant__  unsigned char const_suffix[16];

//struct Lock {
//    int *mutex;
//
//    Lock(void) {
//        int state = 0;
//        hipMalloc((void **) &mutex, sizeof(int));
//        hipMemcpy(mutex, &state, sizeof(int), hipMemcpyHostToDevice);
//    }
//
//    ~Lock(void) {
//        hipFree(mutex);
//    }
//
//    __device__ void lock(void) {
//        while (atomicCAS(mutex, 0, 1) != 0);
//    }
//
//    __device__ void unlock(void) {
//        atomicExch(mutex, 1);
//    }
//};

//__device__ void lock(int *mutex) {
//    while (atomicCAS(mutex, 0, 1) != 0);
//}
//
//__device__ void unlock(int *mutex) {
//    atomicExch(mutex, 0);
//}

//__device__ void lock(int *mutex) {
//    while (atomicCAS(mutex, 0, 1) != 0);
//}
//
//__device__ void unlock(int *mutex) {
//    atomicExch(mutex, 0);
//}

__device__ int device_strlen(const char *s) {

    unsigned int count = 0;
    while (*s != '\0') {
        count++;
        s++;
    }
    return count;
}

__global__ void generate_nonce(unsigned char *nonce, unsigned char *sha, int *dev_mtx, int *dev_stop, int length) {

    int index = threadIdx.x + blockIdx.x * blockDim.x;

    while (!*dev_stop) {

        unsigned char string_aux[1024];
        memcpy(string_aux, const_string, length);

        unsigned char buf[21];
        buf[0] = index % 128;
        buf[1] = (index >> 8) % 128;
        buf[2] = (index >> 16) % 128;
        buf[3] = (index >> 24) % 128;
        buf[4] = '\0';
        memcpy(string_aux + length, buf, device_strlen((const char *) buf));

        unsigned char calculated_sha[21];
        sha1_gpu_context ctx;
        sha1_gpu_starts(&ctx);
        sha1_gpu_update(&ctx, (unsigned char *) string_aux, length + device_strlen((const char *) buf));
        sha1_gpu_finish(&ctx, calculated_sha);
        calculated_sha[20] = '\0';

        if (calculated_sha[15] == const_suffix[0] && calculated_sha[16] == const_suffix[1] &&
            calculated_sha[17] == const_suffix[2] && calculated_sha[18] == const_suffix[3] &&
            calculated_sha[19] == const_suffix[4]) {
            if (atomicCAS(dev_mtx, 0, 1) == 0) {
                *dev_stop = 1;
                memcpy(nonce, buf, device_strlen((const char *) buf) + 1);
                memcpy(sha, calculated_sha, 21);
                atomicExch(dev_mtx, 0);
            }
        }
//        memset(&ctx, 0, sizeof(sha1_gpu_context));
        index += blockDim.x * gridDim.x;
    }
}

int main() {

    unsigned char *str = (unsigned char *) "nonce project g";
    unsigned char *suffix = (unsigned char *) "\x28\x23\x46\x8D\xB0";

    printf("Given string: %s\n", str);
    printf("Given suffix: ");

    for (int i = 0; i < strlen((const char *) suffix); i++) {
        printf("%02X", suffix[i]);
    }
    printf("\n");

    unsigned char nonce[21];
    unsigned char sha[41];

    hipMemcpyToSymbol(HIP_SYMBOL(const_string), str, strlen((const char *) str));
    hipMemcpyToSymbol(HIP_SYMBOL(const_suffix), suffix, strlen((const char *) suffix));

    int *dev_stop;
    int *dev_mtx;
    unsigned char *dev_nonce;
    unsigned char *dev_sha;
    hipError_t cudaStatus;

    cudaStatus = hipSetDevice(0);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipSetDevice failed!");
        goto Error;
    }

    hipMallocManaged(&dev_stop, sizeof(int));
    *dev_stop = 0;

    hipMalloc(&dev_mtx, sizeof(int));
    hipMemset(dev_mtx, 0, sizeof(int));

    cudaStatus = hipMalloc((void **) &dev_nonce, 21);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    cudaStatus = hipMalloc((void **) &dev_sha, 41);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    generate_nonce<<<BLOCKS, THREADS>>>(dev_nonce, dev_sha, dev_mtx, dev_stop, strlen((const char *) str));
    // Errors when lunching the kernel
    cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "Failed to generate nonce: %s\n", hipGetErrorString(cudaStatus));
        goto Error;
    }

    // hipDeviceSynchronize waits for the kernel to finish and returns
    // any errors encountered during the launch.
    cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching generate_nonce!\n", cudaStatus);
        goto Error;
    }

    cudaStatus = hipMemcpy(nonce, dev_nonce, 21, hipMemcpyDeviceToHost);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "Copy nonce from device to host failed!");
        goto Error;
    }

    cudaStatus = hipMemcpy(sha, dev_sha, 41, hipMemcpyDeviceToHost);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "Copy computed sha from device to host failed!");
        goto Error;
    }

    printf("Calculated nonce: %s\n", nonce);
    printf("Calculated nonce bytes: ");
    for (int i = 0; i < strlen((const char *) nonce); i++) {
        printf("%02X", nonce[i]);
    }
    printf("\n");

    printf("Calculated sha: ");
    for (int i = 0; i < strlen((const char *) sha); i++) {
        printf("%02X", sha[i]);
    }
    printf("\n");

    //hipDeviceReset must be  called before exiting in order for profiling and
    // tracing tools such as Nsight and Visual Profiler to show complete traces.
    cudaStatus = hipDeviceReset();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceReset failed!");
        hipFree(dev_stop);
        hipFree(dev_nonce);
        hipFree(dev_sha);
        return 1;
    }
    Error:
    hipFree(dev_stop);
    hipFree(dev_nonce);
    hipFree(dev_sha);
    return 0;
}
